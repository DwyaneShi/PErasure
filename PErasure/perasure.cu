#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "jerasure.h"
#include <algorithm>
extern "C"{
	#include "gf_rand.h"
}

using namespace std;

#define talloc(type, num) (type *) malloc(sizeof(type)*(num))

//__global__ void smpe(int k, int w, int *bitmatrixDevice, int destId, char *dataDevice, char *codingDevice, int dataSize, int numOfLong) {
	//__shared__ char sharedData[dataSize];
	//int blockNumInGrid, threadsPerBlock, threadNumInBlock, tId;
	//blockNumInGrid = blockIdx.x + gridDim.x * blockIdx.y;
	//threadsPerBlock = blockDim.x * blockDim.y;
	//threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	//tId = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	
	//int rowIdx = tId / numOfLong;
	//int colIdx =  tId % numOfLong;
	//int temp = 0;
	//int wordsPerThread = max(1, (dataSize/sizeof(long))/(gridDim.x * gridDim.y * blockDim.x * blockDim.y));
	
	//if(tId * sizeof(long) >= dataSize)
		//return;
	
	//for(dataIdx = 0; dataIdx < k; dataIdx++)
		//memcpy((char *)&sharedData, (char *)(dataDevice + dataIdx * dataSize + tId * ), sizeof(long)); //capire bene cosa succede qui
		////sharedData = *(dataDevice + dataIdx * psize * w)
		//__syncthreads();
		//index = 0;
		//for(i=0; i<w; i++) //qui manca qualcosa 
			//sdIndex = dataIdx + i * psize + colIndex;
			//temp ^= (*(bitmatrixDevice + index) & sharedData[sdIndex]);
			//index++;
		//__syncthreads();
	//codingDevice = 
	
//}

__global__ void gmpe(int k, int w, int *bitmatrixDevice, int destId, long *dataDevice, long *codingDevice, int dataSize, int numOfLong) {
	
	int blockNumInGrid, threadsPerBlock, threadNumInBlock, tId;
	blockNumInGrid = blockIdx.x + gridDim.x * blockIdx.y;
	threadsPerBlock = blockDim.x * blockDim.y;
	threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	tId = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	
	int longIndex = tId % numOfLong;
	int index, dataIdx, i, j;
	long temp;
	long *codPtr, *dtPtr, *innerDtPtr;
	
	if( tId >= numOfLong)
		return;
	
	for(i=0; i<w; i++){
		codPtr = codingDevice + destId * w * numOfLong + i * numOfLong;
		index = destId * k * w * w + i * w;
		temp = 0;
		for(dataIdx=0; dataIdx<k; dataIdx++){
			dtPtr = dataDevice + dataIdx * w * numOfLong;
			for(j=0; j<w; j++){
				if(bitmatrixDevice[index]){
					innerDtPtr = dtPtr + j * numOfLong;
					temp ^= innerDtPtr[longIndex];
				}
				index++;
			}
		}
		codPtr[longIndex] = temp;
	}
}

int main(int argc, char **argv){

	unsigned int m, k, w, i, j, d, r, l, seed, psize;
	unsigned int round;
	//int numBytesBDM, numBytesData, numBytesCoding;
	int *matrix, *bitmatrix, *bitmatrixDevice;
	clock_t start;
	long *data, *dataDevice, *dataTemp, *coding, *codingDevice, *codingTemp;
	dim3 dimGrid(1, 1);
	dim3 dimBlock(16, 1);
    texture<int, 2> texture_reference;
    
    srand(time(NULL));


    if(argc != 5) {
        fprintf(stderr, "Please add arguments k, m, w and size\n");
        exit(1);
    }
	if(sscanf(argv[1], "%d", &k) == 0 || k <= 0) {
		fprintf(stderr, "Wrong m\n");
		exit(1);
	}
	if (sscanf(argv[2], "%d", &m) == 0 || m <= 0) {
		fprintf(stderr, "Wrong k\n");
		exit(1);
	}
	if (sscanf(argv[3], "%d", &w) == 0 || w <= 0 || w > 31) {
		fprintf(stderr, "Wrong w\n");
		exit(1);
	}
	if (sscanf(argv[4], "%d", &psize) == 0) {
		fprintf(stderr, "Wrong packetsize\n");
		exit(1);
	}
	if((k + m) > (1 << w)) {
		fprintf(stderr, "Wrong w, the following must hold: m + k <= 2^w\n");
		exit(1);
	}
	psize = psize/sizeof(long);
//    Creating matrix and BDM
	seed = rand();
	MOA_Seed(seed);
	matrix = talloc(int, m*k);
	for (i = 0; i < m; i++) {
		for (j = 0; j < k; j++) {
			matrix[i*k+j] = galois_single_divide(1, i ^ (m + j), w);
		}
	}

	bitmatrix = jerasure_matrix_to_bitmatrix(k, m, w, matrix);
	//jerasure_print_bitmatrix(bitmatrix, m*w, k*w, w);

//    Generating fake random data
	data = talloc(long , k*w*psize);
	for (i = 0; i < k; i++) {
		//MOA_Fill_Random_Region(data+i*psize*w, psize*w);
		for(j=0; j< w*psize; j++)
			*(data + i*psize*w + j) = 97 + rand()%26;
	}

	coding = talloc(long , m * w * psize);
	
    //	Allocating GPU memory
    start = clock();
    
    hipMalloc(&bitmatrixDevice, m*k*w*w*sizeof(int));
    hipMemcpy(bitmatrixDevice, bitmatrix, m*k*w*w*sizeof(int), hipMemcpyHostToDevice);
    
    //	Compute number of rounds
    size_t free, total;
    hipMemGetInfo(&free, &total);
    
    round = ceil((float)(psize * w * (k + m)) / free);

    printf("Free mem: %lu\n", free);
    
	dataTemp = talloc(long , k * w * (psize/round));
	hipMalloc(&dataDevice, k * w * (psize/round) * sizeof(long));
	
	codingTemp = talloc(long, m * w * (psize/round));
	hipMalloc(&codingDevice, m * w * (psize/round) * sizeof(long));

    for(i = 0; i < round; i++){
	
		// load data chunks

		for(d = 0; d < k; d++)
			for(r = 0; r < w; r++)
				for(l = 0; l < psize / round; l++)
					*(dataTemp + d * w *(psize/round) + r * (psize/round) + l) = *(data + d * w * psize + i * psize/round + r * psize + l);

		hipMemcpy(dataDevice, dataTemp, k * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);
		hipMemcpy(codingDevice, codingTemp, m * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);

		for(j = 0; j < m; j++)
			//smpe<<<dimGrid, dimBlock>>>(k, w, bitmatrixDevice + j * w * w * k, j, dataDevice, codingDevice, (psize/round) * w, sizeof(long));
			gmpe<<<dimGrid, dimBlock>>>(k, w, bitmatrixDevice + j * w * w * k, j, dataDevice, codingDevice, (psize/round) * w, (psize/round));
		// copy coding back to main memory
		
		hipMemcpy(codingTemp, codingDevice, m * w * (psize/round) * sizeof(long), hipMemcpyDeviceToHost);
		// Extend_Coding_Device(codingTemp, coding, destId);
		

		hipFree(dataDevice);
		hipFree(codingDevice);
	}
    printf("Encoding complete, time elapsed: %.2fs\n", (clock() - (float)start) / CLOCKS_PER_SEC);
    
    for(i = 0; i < k; i++){
		for(j = 0; j < w * psize; j++)
			printf("%ld ", *(dataTemp+i*w*psize + j));
		printf("\n");
	}
	printf("\n");
	
	for(i = 0; i < m; i++){
		for(j = 0; j < w * psize; j++)
			printf("%ld ", *(codingTemp + i*w*psize + j));
		printf("\n");
	}
	printf("\n");

    return 0;
}
