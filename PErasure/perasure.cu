#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "jerasure.h"
extern "C"{
	#include "gf_rand.h"
}

using namespace std;

#define talloc(type, num) (type *) malloc(sizeof(type)*(num))

__global__ void smpe(int k, int w, char **dataDevice, char **codingDevice, int psize, int numOfLong)
{
	// TO DO
	int i=0;
	i += 1;
}

int main(int argc, char **argv){
	
	unsigned int m, k, w, i, j, d, r, l, seed, psize;
	unsigned int round;
	int numBytesBDM, numBytesData, numBytesCoding;
	int *matrix, *bitmatrix, *bitmatrixDevice;
	clock_t start;
	char **data, **dataDevice, **dataTemp, **coding, **codingDevice, **codingTemp;
	dim3 dimBlock(4, 4);
    dim3 dimGrid(4, 4);
    texture<int, 2> texture_reference;
    
    
    if(argc != 5) {
        fprintf(stderr, "Please add arguments m, k, w and size\n");
        exit(1);
    }
	if(sscanf(argv[1], "%d", &k) == 0 || k <= 0) {
		fprintf(stderr, "Wrong m\n"); 
		exit(1);
	}
	if (sscanf(argv[2], "%d", &m) == 0 || m <= 0) {
		fprintf(stderr, "Wrong k\n"); 
		exit(1);
	}
	if (sscanf(argv[3], "%d", &w) == 0 || w <= 0 || w > 31) {
		fprintf(stderr, "Wrong w\n"); 
		exit(1);
	}
	if (sscanf(argv[4], "%d", &psize) == 0) {
		fprintf(stderr, "Wrong packetsize\n"); 
		exit(1);
	}	
	if((k + m) > (1 << w)) {
		fprintf(stderr, "Wrong w, the following must hold: m + k <= 2^w\n"); 
		exit(1);
	}
   
//    Creating matrix and BDM
	seed = rand();
	MOA_Seed(seed);
	matrix = talloc(int, m*k);
	for (i = 0; i < m; i++) {
		for (j = 0; j < k; j++) {
			matrix[i*k+j] = galois_single_divide(1, i ^ (m + j), w);
		}
	}
	
	bitmatrix = jerasure_matrix_to_bitmatrix(k, m, w, matrix);
	
//    Generating fake random data		
	data = talloc(char *, k);
	for (i = 0; i < k; i++) {
		data[i] = talloc(char, psize*w);
		MOA_Fill_Random_Region(data[i], psize*w);
	}

	coding = talloc(char *, m);
	for (i = 0; i < m; i++) {
		coding[i] = talloc(char, psize*w);
	}
    
    //	Allocating GPU memory   
    start = clock();
    numBytesBDM = (m * k) * sizeof(int);
    hipMalloc(&bitmatrixDevice, numBytesBDM);
	hipMemcpy(bitmatrixDevice, bitmatrix, numBytesBDM, hipMemcpyHostToDevice);
    hipBindTexture(NULL, texture_reference, bitmatrixDevice, numBytesBDM);
    
    numBytesData = k * sizeof(long);
    hipMalloc(&dataDevice, numBytesData);
    
    numBytesCoding = m * sizeof(long);
    hipMalloc(&codingDevice, numBytesCoding);
    hipMemcpy(codingDevice, coding, numBytesCoding, hipMemcpyHostToDevice);
    
    //	Compute number of rounds
    size_t free, total;
    hipMemGetInfo(&free, &total);
    round = ceil((float)(psize * w * (k + m)) / free);
    dataTemp = talloc(char *, k);
	for (i = 0; i < k; i++)
		dataTemp[i] = talloc(char, psize/round*w);
	codingTemp = talloc(char *, m);
	for (i = 0; i < m; i++)
		codingTemp[i] = talloc(char, psize/round*w);
    printf("Free mem: %lu\n", free);
    
    for(i=0; i < round; i++){

		// load data chunks
		for(d=0; d < k; d++)
			for(r=0; r < w; r++)
				for(l=0; l < psize / round; l++)
					dataTemp[d][r*psize/round + l] += data[d][r*psize + l + i*psize/round];

		hipMemcpy(dataDevice, dataTemp, numBytesData, hipMemcpyHostToDevice);

		for(j=0; j < m; j++)
			smpe<<<dimGrid, dimBlock>>>(k, w, dataDevice, codingTemp+(m*w*psize/round), psize, sizeof(long));
		// copy coding back to main memory
		hipMemcpy(codingTemp, codingDevice, numBytesCoding, hipMemcpyDeviceToHost);
		// Extend_Coding_Device(codingTemp, coding, destId);

		hipFree(dataDevice);
		hipFree(codingDevice);
	}
    printf("Encoding complete, time elapsed: %.2fs\n", (clock() - (float)start) / CLOCKS_PER_SEC);

    return 0;
}
