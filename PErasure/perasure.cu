#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "jerasure.h"
#include <algorithm>
extern "C"{
	#include "gf_rand.h"
}

using namespace std;

#define talloc(type, num) (type *) malloc(sizeof(type)*(num))

//__global__ void smpe(int k, int w, int *bitmatrixDevice, int destId, char *dataDevice, char *codingDevice, int dataSize, int numOfLong) {
	//__shared__ char sharedData[dataSize];
	//int blockNumInGrid, threadsPerBlock, threadNumInBlock, tId;
	//blockNumInGrid = blockIdx.x + gridDim.x * blockIdx.y;
	//threadsPerBlock = blockDim.x * blockDim.y;
	//threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	//tId = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	
	
	//if(tId * sizeof(long) >= dataSize)
		//return;
	
	//for(dataIdx = 0; dataIdx < k; dataIdx++)
		//memcpy((char *)&sharedData, (char *)(dataDevice + dataIdx * dataSize + tId * ), sizeof(long));
		////sharedData = *(dataDevice + dataIdx * psize * w)
		//__syncthreads();
		//for(i=0; i<w; i++) 
			//sdIndex = dataIdx + i * psize + colIndex;
			//temp ^= (*(bitmatrixDevice + index) & sharedData[sdIndex]);
			//index++;
		//__syncthreads();
	//codingDevice = 
	
//}

__global__ void gmpe(int k, int w, int *bitmatrixDevice, int destId, long *dataDevice, long *codingDevice, int dataSize, int numOfLong) {
	
	int blockNumInGrid, threadsPerBlock, threadNumInBlock, tId;
	blockNumInGrid = blockIdx.x + gridDim.x * blockIdx.y;
	threadsPerBlock = blockDim.x * blockDim.y;
	threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	tId = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	
	int longIndex = tId % numOfLong;
	int index, dataIdx, i, j;
	long temp;
	long *codPtr, *dtPtr, *innerDtPtr;
	
	if( tId >= numOfLong)
		return;
	
	for(i=0; i<w; i++){
		codPtr = codingDevice + destId * w * numOfLong + i * numOfLong;
		index = destId * k * w * w + i * k * w;
		temp = 0;
		for(dataIdx=0; dataIdx<k; dataIdx++){
			dtPtr = dataDevice + dataIdx * w * numOfLong;
			for(j=0; j<w; j++){
				if(bitmatrixDevice[index]){
					innerDtPtr = dtPtr + j * numOfLong;
					temp ^= innerDtPtr[longIndex];
				}
				index++;
			}
		}
		codPtr[longIndex] = temp;
	}
}


void extendCodingDevice(long *codingTemp, long *coding, int i, int m, int psize, int offset, int rows){
	int k, j;
	
	for(k=0; k<m; k++)	
		for(j=0; j<rows; j++)
			memcpy((coding + k * psize * rows + psize * j + i * offset), (codingTemp + k * offset * rows + offset * j), sizeof(long) * offset);
}

int main(int argc, char **argv){

	unsigned int m, k, w, i, j, d, r, l, y, seed, psize;
	unsigned int round;
	//int numBytesBDM, numBytesData, numBytesCoding;
	int *matrix, *bitmatrix, *bitmatrixDevice;
	clock_t start;
	long *data, *dataDevice, *dataTemp, *coding, *codingDevice, *codingTemp;
    
    srand(time(NULL));


    if(argc != 5) {
        fprintf(stderr, "Please add arguments k, m, w and size\n");
        exit(1);
    }
	if(sscanf(argv[1], "%d", &k) == 0 || k <= 0) {
		fprintf(stderr, "Wrong m\n");
		exit(1);
	}
	if (sscanf(argv[2], "%d", &m) == 0 || m <= 0) {
		fprintf(stderr, "Wrong k\n");
		exit(1);
	}
	if (sscanf(argv[3], "%d", &w) == 0 || w <= 0 || w > 31) {
		fprintf(stderr, "Wrong w\n");
		exit(1);
	}
	if (sscanf(argv[4], "%d", &psize) == 0) {
		fprintf(stderr, "Wrong packetsize\n");
		exit(1);
	}
	if((k + m) > (1 << w)) {
		fprintf(stderr, "Wrong w, the following must hold: m + k <= 2^w\n");
		exit(1);
	}
	psize = psize/sizeof(long);
	
	dim3 dimGrid(1, 1);
	dim3 dimBlock(psize, 1);
	
//    Creating matrix and BDM
	seed = rand();
	MOA_Seed(seed);
	matrix = talloc(int, m*k);
	for (i = 0; i < m; i++) {
		for (j = 0; j < k; j++) {
			matrix[i*k+j] = galois_single_divide(1, i ^ (m + j), w);
		}
	}

	bitmatrix = jerasure_matrix_to_bitmatrix(k, m, w, matrix);
	//jerasure_print_bitmatrix(bitmatrix, m*w, k*w, w);

//    Generating fake random data
	data = talloc(long , k*w*psize);
	for (i = 0; i < k; i++) {
		for(j=0; j< w*psize; j++)
			*(data + i*psize*w + j) = 97 + rand()%26;
	}

	coding = talloc(long , m * w * psize);
	
    //	Allocating GPU memory
    start = clock();
    
    hipMalloc(&bitmatrixDevice, m*k*w*w*sizeof(int));
    hipMemcpy(bitmatrixDevice, bitmatrix, m*k*w*w*sizeof(int), hipMemcpyHostToDevice);
    
    //	Compute number of rounds
    size_t free, total;
    hipMemGetInfo(&free, &total);
    
    round = ceil((float)(psize * w * (k + m)) / free);

    printf("Free mem: %lu\n", free);
    
	dataTemp = talloc(long , k * w * (psize/round));
	hipMalloc(&dataDevice, k * w * (psize/round) * sizeof(long));
	
	codingTemp = talloc(long, m * w * (psize/round));
	hipMalloc(&codingDevice, m * w * (psize/round) * sizeof(long));
	
    for(i = 0; i < round; i++){
	
		// load data chunks
		for(d = 0; d < k; d++)
			for(r = 0; r < w; r++)
				for(l = 0; l < psize / round; l++)
					*(dataTemp + d * w *(psize/round) + r * (psize/round) + l) = *(data + d * w * psize + i * psize/round + r * psize + l);
					
		hipMalloc(&dataDevice, k * w * (psize/round) * sizeof(long));
		hipMalloc(&codingDevice, m * w * (psize/round) * sizeof(long));
		hipMemcpy(dataDevice, dataTemp, k * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);
		hipMemcpy(codingDevice, codingTemp, m * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);

		for(j = 0; j < m; j++)
			//smpe<<<dimGrid, dimBlock>>>(k, w, bitmatrixDevice + j * w * w * k, j, dataDevice, codingDevice, (psize/round) * w, sizeof(long));
			gmpe<<<dimGrid, dimBlock>>>(k, w, bitmatrixDevice, j, dataDevice, codingDevice, (psize/round) * w, (psize/round));
			
		// copy coding back to main memory
		hipDeviceSynchronize();
		hipMemcpy(codingTemp, codingDevice, m * w * (psize/round) * sizeof(long), hipMemcpyDeviceToHost);
		extendCodingDevice(codingTemp, coding, i, m, psize, (psize/round), w);
		
		hipFree(dataDevice);
		hipFree(codingDevice);
	}
    printf("Encoding complete, time elapsed: %.2fs\n", (clock() - (float)start) / CLOCKS_PER_SEC);
    
    for(i = 0; i < k; i++){
		for(j = 0; j < w * psize; j++)
			printf("%c ", (char)*(data + i*w*psize + j));
		printf("\n");
	}
	printf("\n");
	
	for(i = 0; i < m; i++){
		for(j = 0; j < w * psize; j++)
			printf("%c ", (char)*(coding + i*w*psize + j));
		printf("\n");
	}
	printf("\n");

    return 0;
}
