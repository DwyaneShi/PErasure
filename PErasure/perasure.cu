#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "jerasure.h"
extern "C"{
	#include "gf_rand.h"
}

using namespace std;

#define talloc(type, num) (type *) malloc(sizeof(type)*(num))

texture<int, 1, hipReadModeElementType> texBDM;

__global__ void gmpe(int k, int w, int destId, long *dataDevice, long *codingDevice, int numOfLong) {
	
	int blockNumInGrid, threadsPerBlock, threadNumInBlock, tId;
	blockNumInGrid = blockIdx.x + gridDim.x * blockIdx.y;
	threadsPerBlock = blockDim.x * blockDim.y;
	threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	tId = blockNumInGrid * threadsPerBlock + threadNumInBlock;
	
	int longIndex = tId % numOfLong;
	int index, dataIdx, i, j;
	long temp;
	long *codPtr, *dtPtr, *innerDtPtr;
	
	if( tId >= numOfLong)
		return;
	
	for(i=0; i<w; i++){
		codPtr = codingDevice + destId * w * numOfLong + i * numOfLong;
		index = destId * k * w * w + i * k * w;
		temp = 0;
		for(dataIdx=0; dataIdx<k; dataIdx++){
			dtPtr = dataDevice + dataIdx * w * numOfLong;
			for(j=0; j<w; j++){
				if(tex1Dfetch(texBDM, index)){
					innerDtPtr = dtPtr + j * numOfLong;
					temp ^= innerDtPtr[longIndex];
				}
				index++;
			}
		}
		codPtr[longIndex] = temp;
	}
}


void extendCodingDevice(long *codingTemp, long *coding, int i, int m, int psize, int offset, int rows){
	int k, j;
	
	for(k=0; k<m; k++)	
		for(j=0; j<rows; j++)
			memcpy((coding + k * psize * rows + psize * j + i * offset), (codingTemp + k * offset * rows + offset * j), sizeof(long) * offset);
}

int main(int argc, char **argv){

	unsigned int m, k, w, i, j, d, r, seed, psize, round;
	int *matrix, *bitmatrix, *bitmatrixDevice;
	clock_t start;
	long *data, *dataDevice, *dataTemp, *coding, *codingDevice, *codingTemp;
    
    texBDM.filterMode = hipFilterModePoint;
    texBDM.addressMode[0] = hipAddressModeClamp;
    
    srand(time(NULL));
    seed = rand();
	MOA_Seed(seed);

    if(argc != 5) {
        fprintf(stderr, "Please add arguments k, m, w and size\n");
        exit(1);
    }
	if(sscanf(argv[1], "%d", &k) == 0 || k <= 0) {
		fprintf(stderr, "Wrong k. It must be strictly postive.\n");
		exit(1);
	}
	if (sscanf(argv[2], "%d", &m) == 0 || m <= 0) {
		fprintf(stderr, "Wrong m. It must be strictly positive.\n");
		exit(1);
	}
	if (sscanf(argv[3], "%d", &w) == 0 || w <= 0 || w > 31) {
		fprintf(stderr, "Wrong w. It must be between 0 and 32.\n");
		exit(1);
	}
	if (sscanf(argv[4], "%d", &psize) == 0){// || psize%sizeof(long) != 0) {
		fprintf(stderr, "Wrong packetsize. It must be an amount of bytes multiple of long.\n");
		exit(1);
	}
	if((k + m) > (1 << w)) {
		fprintf(stderr, "Wrong combinatio of k, m and w. The following must hold: m + k <= 2^w\n");
		exit(1);
	}
	psize = psize/sizeof(long);
	
	//int dimG = min((int)ceil(sqrt(psize)), 1024);
	//int dimB = min((int)ceil((float)psize/(dimG*dimG)), 1024);
	//dim3 dimGrid(dimG, dimG);
	//dim3 dimBlock(dimB, dimB);	
	
	
	int threadPerBlock = min(psize, 1024);
	int nBlocks = ceil((float)psize/threadPerBlock);
	
//    Creating CRS matrix and BDM

	matrix = talloc(int, m*k);
	for (i = 0; i < m; i++) {
		for (j = 0; j < k; j++) {
			matrix[i*k+j] = galois_single_divide(1, i ^ (m + j), w);
		}
	}

	bitmatrix = jerasure_matrix_to_bitmatrix(k, m, w, matrix);

//	Generating fake random data

	data = talloc(long , k*w*psize);
	for (i = 0; i < k; i++) {
		for(j=0; j< w*psize; j++)
			*(data + i*psize*w + j) = 97 + rand()%26;
	}
	
//	Allocating space for coding devices

	coding = talloc(long , m * w * psize);
	
//	Allocating GPU memory
    
    hipMalloc(&bitmatrixDevice, m*k*w*w*sizeof(int));
    hipMemcpy(bitmatrixDevice, bitmatrix, m*k*w*w*sizeof(int), hipMemcpyHostToDevice);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
    hipBindTexture(0, texBDM, bitmatrixDevice, channelDesc, m*k*w*w*sizeof(int));
    
//	Computing number of rounds
    
    size_t free, total;
    hipMemGetInfo(&free, &total);
    
    round = ceil((float)(psize * w * (k + m) * sizeof(long)) / free);

    dataTemp = talloc(long , k * w * (psize/round));
	
	codingTemp = talloc(long, m * w * (psize/round));


    for(i = 0; i < round; i++){
		
		// load data chunks when needed
		if(round > 1){
			for(d = 0; d < k; d++)
				for(r = 0; r < w; r++)
					memcpy((dataTemp + d * w *(psize/round) + r * (psize/round)), (data + d * w * psize + i * psize/round + r * psize), sizeof(long) * (psize/round));
											
			hipMalloc(&dataDevice, k * w * (psize/round) * sizeof(long));
			hipMalloc(&codingDevice, m * w * (psize/round) * sizeof(long));
			hipMemcpy(dataDevice, dataTemp, k * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);
			hipMemcpy(codingDevice, coding, m * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);
		} //else load all the data
		else{
			hipMalloc(&dataDevice, k * w * (psize/round) * sizeof(long));
			hipMalloc(&codingDevice, m * w * (psize/round) * sizeof(long));
			hipMemcpy(dataDevice, data, k * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);
			hipMemcpy(codingDevice, codingTemp, m * w * (psize/round) * sizeof(long), hipMemcpyHostToDevice);
		}
		
		start = clock();
		for(j = 0; j < m; j++)
			gmpe<<<nBlocks, threadPerBlock>>>(k, w, j, dataDevice, codingDevice, (psize/round));
			
		// copy coding back to main memory
		hipDeviceSynchronize();
		printf("Encoding complete, time elapsed: %.8fs\n", (clock() - (float)start) / CLOCKS_PER_SEC);

		hipMemcpy(codingTemp, codingDevice, m * w * (psize/round) * sizeof(long), hipMemcpyDeviceToHost);
		extendCodingDevice(codingTemp, coding, i, m, psize, (psize/round), w);

		hipFree(dataDevice);
		hipFree(codingDevice);
	}
    
    hipUnbindTexture(texBDM);
    //Status after coding
    //for(i = 0; i < k; i++){
		//for(j = 0; j < w * psize; j++)
			//printf("%02x ", (unsigned char)*(data + i*w*psize + j));
		//printf("\n");
	//}
	//printf("\n");
	
	//for(i = 0; i < m; i++){
		//for(j = 0; j < w * psize; j++)
			//printf("%02x ", (unsigned char)*(coding + i*w*psize + j));
		//printf("\n");
	//}
	//printf("\n");
    
    //// Erasing random m devices
    //int random[m+1];
    //bool flag;
    //for(i = 0; i < m;) {
        //r = MOA_Random_W(w, 1) % (k + m);
        //flag = true;
        //for (j = 0; j < m; j++)
            //if (r == random[j]) flag = false;
        //if (flag) {
            //random[i] = r;
            //i++;
        //}
    //}
    //random[i] = -1;
    //for(i = 0; i < m; i++) {
        //if (random[i] < k)
            //bzero((data + random[i] * w * psize), w*psize * sizeof(long));
        //else bzero((coding + (random[i] - k) * w * psize), w*psize * sizeof(long));
    //}
    //printf("Erased %d random devices\n", m);
       
    //for(i = 0; i < k; i++){
		//for(j = 0; j < w * psize; j++)
			//printf("%02x ", (unsigned char)*(data + i*w*psize + j));
		//printf("\n");
	//}
	//printf("\n");
	
	//for(i = 0; i < m; i++){
		//for(j = 0; j < w * psize; j++)
			//printf("%02x ", (unsigned char)*(coding + i*w*psize + j));
		//printf("\n");
	//}
	//printf("\n");
	
	//char **data2, **coding2;
	
	//data2 = talloc(char *, k);
	//for (i = 0; i < k; i++) {
		//data2[i] = talloc(char, psize*w);
	//}

	//coding2 = talloc(char *, m);
	//for (i = 0; i < m; i++) {
		//coding2[i] = talloc(char, psize*w);
	//}
	
	//for(i = 0; i < k; i++){
		//for(j = 0; j < w * psize; j++)
			//data2[i][j] = (char)*(data + i*w*psize + j);
	//}
	
	//for(i = 0; i < m; i++){
		//for(j = 0; j < w * psize; j++)
			//coding2[i][j] = (char)*(coding + i*w*psize + j);
	//}
	
	//start = clock();
	//jerasure_bitmatrix_decode(k, m, w, bitmatrix, 0, random, data2, coding2, w*psize, psize);
	//printf("Devices recovered, time elapsed: %.4fs\n", (clock() - (float)start) / CLOCKS_PER_SEC);
	
	//for(i = 0; i < k; i++){
		//for(j = 0; j < w * psize; j++)
			//printf("%02x ", (unsigned char)data2[i][j]);
		//printf("\n");
	//}
	//printf("\n");
	
	//for(i = 0; i < m; i++){
		//for(j = 0; j < w * psize; j++)
			//printf("%02x ", (unsigned char)coding2[i][j]);
		//printf("\n");
	//}
	//printf("\n");

    return 0;
}
